#include "hip/hip_runtime.h"
#include "cppdefs.h"
#include "cuda_kernels.cuh"

EXTERN __global__ void rotation90_kernel_cuda(unsigned* image, unsigned* alt_image, unsigned DIM) {
	unsigned i = get_i();
	unsigned j = get_j();
	next_img(DIM - i - 1, j) = cur_img(j, i);
}